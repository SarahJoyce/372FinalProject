#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define INTERVALS 5000000000
#define nblocks 10
#define nthreads 10

double num_pi = 0.0;

__global__ void pi(double *area, int threads, int blocks){
  //do individual thread stuff
  double xi;
  int i;
  int threadindex = threadIdx.x + blockIdx.x*threads;
  for (i=threadindex; i<INTERVALS; i+=threads*blocks) {
    xi=(1.0/INTERVALS)*(i+0.5);
    area[i] += 4.0/(INTERVALS*(1.0+xi*xi));
  }
}

int main(int argc, char **argv) {
  clock_t start_time = clock();

  //nblocks = (int)atoi(argv[1]);
  //nthreads = (int)atoi(argv[2]);

  dim3 numBlocks(nblocks);
  dim3 threadsPerBlock(nthreads);

  double *area;
  double *d_area;
  area = (double *)malloc(nblocks*nthreads*sizeof(double));
  for(int i=0; i<nblocks*nthreads; i++){
    area[i]=0;
  }

  hipMalloc((void **) &d_area, nblocks*nthreads*sizeof(double));

  hipMemcpy(d_area, area, nblocks*nthreads*sizeof(double), hipMemcpyHostToDevice);

  pi<<<numBlocks, threadsPerBlock>>>(d_area, nthreads, nblocks);

  hipMemcpy(&area, d_area, nblocks*nthreads*sizeof(double), hipMemcpyDeviceToHost);

  //add everything together
  for(int i=0; i<nblocks*nthreads; i++){
    num_pi += area[i];
  }
  num_pi = num_pi * (1.0/INTERVALS);

  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  
  printf("Pi is %.2f (nthreads = %d, time = %f sec.)\n", num_pi, nthreads, time);
  
  free(area);
  hipFree(d_area);
  fflush(stdout);
  return 0;
}
