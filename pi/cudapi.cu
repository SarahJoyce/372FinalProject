#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define INTERVALS 5000000000L

//int rank;
//int numprocs;
long double pi;

__global__ void pi(void){
//do individual thread stuff
}

//int main(void)??
int main(int argc, char *argv[]) {    
  long double area = 0.0;
  long double xi;
  long i;

  clock_t start_time = clock();

  //fix rank and numprocs below
  for (i=(long)rank; i<INTERVALS; i+=(long)numprocs) {
    xi=(1.0L/INTERVALS)*(i+0.5L);
    area += 4.0L/(INTERVALS*(1.0L+xi*xi));
  }
  //MPI_Reduce used to be here (add everything together)
  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  //back to main here (used to be rank == 0)
  printf("Pi is %20.17Lf (nprocs = %d, time = %f sec.)\n", pi, numprocs, time);
  fflush(stdout);
  return 0;
}
