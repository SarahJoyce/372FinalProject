#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define INTERVALS 5000000000

int nthreads;
int nblocks;
double num_pi;

__global__ void pi(double *d_area){
  //do individual thread stuff
  double xi;
  long i;
  int a = 0;
  int threadindex = threadIdx.x + blockIdx.x*blockDim.x;
  int threads = gridDim.x * blockDim.x;
  for (i=threadindex; i<INTERVALS; i+=threads) {
    xi=(1.0/INTERVALS)*(i+0.5);
    a = 4.0/(INTERVALS*(1.0+xi*xi));
  }d_area[threadindex] = a;
}

int main(int argc, char **argv) {
  clock_t start_time = clock();

  nblocks = (int)atoi(argv[1]);
  nthreads = (int)atoi(argv[2]);

  double *area;
  double *d_area;
  area = (double *)malloc(nblocks*nthreads*sizeof(double));
  for(int i=0; i<nblocks*nthreads; i++){
    area[i]=0;
  }

  hipMalloc((double **) &d_area, nblocks*nthreads*sizeof(double));

  hipMemcpy(d_area, area, nblocks*nthreads*sizeof(double), hipMemcpyHostToDevice);

  pi<<<nblocks, nthreads>>>(d_area);

  hipMemcpy(area, d_area, nblocks*nthreads*sizeof(double), hipMemcpyDeviceToHost);

  //add everything together
  for(int i=0; i<nblocks*nthreads; i++){
    num_pi += area[i];
  }

  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  
  printf("Pi is %.2f (nthreads = %d, time = %f sec.)\n", num_pi, nthreads, time);
  
  free(area);
  hipFree(d_area);
  fflush(stdout);
  return 0;
}
