#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define INTERVALS 5000000000L

int nthreads;
int nblocks;
double num_pi;

__global__ void pi(double *area, int threads, int blocks){
  //do individual thread stuff
  double xi;
  int i;
  int threadindex = threadIdx.x + blockIdx.x*threads;
  for (i=threadindex; i<INTERVALS; i+=threads*blocks) {
    xi=(1.0/INTERVALS)*(i+0.5);
    area[i] += 4.0/(INTERVALS*(1.0+xi*xi));
  }
}

int main(int argc, char **argv) {
  clock_t start_time = clock();

  nthreads = (int)atoi(argv[1]);
  nblocks = (int)atoi(argv[2]);

  double *area;
  double *d_area;
  area = (double *)malloc(sizeof(double));

  hipMalloc((void **) &d_area, nblocks*nthreads*sizeof(double));

  hipMemcpy(d_area, &area, nblocks*nthreads*sizeof(double), hipMemcpyHostToDevice);
  //fix the line below
  pi<<<nblocks, nthreads>>>(d_area, nthreads, nblocks);

  hipMemcpy(&area, d_area, nblocks*nthreads*sizeof(double), hipMemcpyDeviceToHost);

  //add everything together
  for(int i=0; i<nblocks*nthreads; i++){
    num_pi+=area[i];
  }

  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  
  printf("Pi is %.2f (nthreads = %d, time = %f sec.)\n", num_pi, nthreads, time);
  
  free(area);
  hipFree(d_area);
  fflush(stdout);
  return 0;
}
