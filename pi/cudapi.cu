#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define INTERVALS 5000000000L

int nthreads;
int nblocks;
double num_pi = 0.0;

__global__ void pi(double *area, int threads, int blocks){
  //do individual thread stuff
  double xi;
  int i;
  int threadindex = threadIdx.x + blockIdx.x*threads;
  for (i=threadindex; i<INTERVALS; i+=threads*blocks) {
    xi=(1.0/INTERVALS)*(i+0.5);
    area[i] += 4.0/(INTERVALS*(1.0+xi*xi));
  }
}

int main(int argc, char **argv) {
  assert(argc==3);

  int arg1 = (int)atoi(argv[1]);
  int arg2 = (int)atoi(argv[2]);
  //int arg3 = (int)atoi(argv[3]);

  printf("Arg1: %d", arg1);
  printf("Arg2: %d", arg2);
  //printf("Arg3: %d", arg3);

  clock_t start_time = clock();

  nblocks = (int)atoi(argv[1]);
  nthreads = (int)atoi(argv[2]);

  dim3 numBlocks(nblocks, 1, 1);
  dim3 threadsPerBlock(nthreads, 1, 1);

  double *area;
  double *d_area;
  area = (double *)malloc(sizeof(double));
  for(int i=0; i<nblocks*nthreads; i++){
    area[i] = 0;
  }

  hipMalloc((void **) &d_area, nblocks*nthreads*sizeof(double));

  hipMemcpy(d_area, &area, nblocks*nthreads*sizeof(double), hipMemcpyHostToDevice);

  pi<<<numBlocks, threadsPerBlock>>>(d_area, nthreads, nblocks);
  
  hipDeviceSynchronize();

  hipMemcpy(&area, d_area, nblocks*nthreads*sizeof(double), hipMemcpyDeviceToHost);

  //add everything together
  for(int i=0; i<nblocks*nthreads; i++){
    num_pi = (num_pi + area[i])*(1.0/INTERVALS);
  }

  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  
  printf("Pi is %.2f (nthreads = %d, time = %f sec.)\n", num_pi, nthreads, time);
  
  free(area);
  hipFree(d_area);
  fflush(stdout);
  return 0;
}
