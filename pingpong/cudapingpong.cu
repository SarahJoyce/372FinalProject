#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define NPINGS 1000000

__global__ void kernel(void){
}

int main(int argc, char *argv[]) {
  //change the params for the line below later
  kernel<<<1,1>>>();	
  clock_t start_time, finish_time, total_time;	
  
  for(int i=1; i<numprocs; i++){
    if(rank == 0){
	start_time = clock();
	for(int j=0; j<NPINGS; j++){
	  //MPI_Send(NULL, 0, MPI_CHAR, i, 99, MPI_COMM_WORLD);
	  //MPI_Recv(NULL, 0, MPI_CHAR, i, 99, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	}
    }else if(rank == i){
	for(int j=0; j<NPINGS; j++){
	  //MPI_Recv(NULL, 0, MPI_CHAR, 0, 99, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	  //MPI_Send(NULL, 0, MPI_CHAR, 0, 99, MPI_COMM_WORLD);
	}
    }
    finish_time = clock();
    total_time =(double)((finish_time-start_time)/(CLOCKS_PER_SEC)(2*NPINGS);
    printf("Average time to transmit between 0 and %d: %11.10f\n", i, total_time);
	fflush(stdout);
  }
  return 0;
}
