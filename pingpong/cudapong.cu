#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void pingpong(void){}

int main(int argc, char *argv[]){
	int runs=atoi(argv[1]);

	clock_t start_time=clock();
	for(int i=0;i<runs;i++){
	}
	clock_t finish_time=clock();
	
	clock_t cuda_start=clock();
	for(int i=0;i<runs;i++){
		pingpong<<<1,1>>>();
	}
	clock_t cuda_finish=clock();

	double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
	time=time/runs;
	double cuda_time=(double)(cuda_finish-cuda_start)/CLOCKS_PER_SEC;
	cuda_time=cuda_time/runs;
	printf("Each loop took an average of %11.10f sec without cuda.\n",time);
	printf("Each loop took an average of %11.10f sec with cuda.\n",cuda_time);
}
