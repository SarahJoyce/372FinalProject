#include "hip/hip_runtime.h"
/* 
 * Some tests:
 * 1 -> 0
 * 2 -> 7080
 * 100 -> 0076
 *
 * 100000000 takes 4.2 seconds on my MacBook Air
 * 1000000000  takes 43 seconds on same machine
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//int rank;
//int numprocs;
int pin;

__global__ void pin(void){
}

//int main(void)??
int main(int argc, char *argv[]) {
  //change the params below later
  pin<<<1,1>>>();
  clock_t start_time = clock();
  assert(argc==2);

  double stop = (double)atol(argv[1]);
  assert(stop >= 1.0);

  int result = 0;

  //fix rank and numprocs below
  for (double x = (double)rank; x < stop; x += (double)numprocs) {
    double tmp = sin(x);
    double tmp2 = tmp*tmp;
    int z = (int)(tmp2*10000.0);

    result = (result + z)%10000; // 0<=result<10000
  }
  //MPI_Reduce used to be here
  pin = pin%10000;
  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  printf("The PIN is %d (nprocs = %d, time = %f sec.)\n", pin, numprocs, time);
  fflush(stdout);
  return 0;
}
