#include "hip/hip_runtime.h"
/* 
 * Some tests:
 * 1 -> 0
 * 2 -> 7080
 * 100 -> 0076
 *
 * 100000000 takes 4.2 seconds on my MacBook Air
 * 1000000000  takes 43 seconds on same machine
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//int rank;
//int numprocs;
__global__ void pin(int *d_result, double stop){
  int i=blockDim.x * blockIdx.x + threadIdx.x;
  double x=(double) i;
  for(x;x<(int)stop;x+=(double)(gridDim.x*blockDim.x)){
    double tmp=sin(x);
    tmp=tmp*tmp;
    int z=(int) (tmp*10000.0);
    d_result[i]=(d_result[i]+z)%10000;
//    printf("i=%d is %d\n",i,d_result[i]);
  }
}

//int main(void)??
int main(int argc, char *argv[]) {
  //change the params below later
  int numBlocks=100;
  int numThreads=100;
  clock_t start_time = clock();
  assert(argc==2);

  double stop = (double)atol(argv[1]);
  assert(stop >= 1.0);

  int *result = (int*)malloc(numBlocks*numThreads*sizeof(int));
  for(int i=0;i<numBlocks*numThreads;i++){
    result[i]=0;
  }


  int *d_result;
  hipMalloc((void**)&d_result, numBlocks*numThreads*sizeof(int));
  hipMemcpy(d_result, result, numBlocks*numThreads*sizeof(int), hipMemcpyHostToDevice);

  pin<<<numBlocks,numThreads>>>(d_result,stop);

  //MPI_Reduce used to be here
  hipMemcpy(result, d_result, numBlocks*numThreads*sizeof(int),hipMemcpyDeviceToHost);
  
  int pin=0;

  for(int i=0;i<numBlocks*numThreads;i++){
    pin=(pin+result[i])%10000;
//    printf("result %d is %d.\n",i,result[i]);
  }
  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  printf("The PIN is %d (numBlocks = %d, numThreads = %d, time = %f sec.)\n", pin, numBlocks, numThreads, time);
  hipFree(d_result);
  free(result);
  fflush(stdout);
  return 0;
}
