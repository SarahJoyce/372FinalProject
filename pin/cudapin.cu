#include "hip/hip_runtime.h"
/* 
 * Some tests:
 * 1 -> 0
 * 2 -> 7080
 * 100 -> 0076
 *
 * 100000000 takes 4.2 seconds on my MacBook Air
 * 1000000000  takes 43 seconds on same machine
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//int rank;
//int numprocs;
__global__ void pin(int *d_result, double stop, int numBlocks){
  int i=blockIdx.x;
  double x=(double) i;
  while(x<(int)stop){
    double tmp=sin(x);
    tmp=tmp*tmp;
    int z=(int) (tmp*10000.0);
    d_result[i]=(d_result[i]+z)%10000;
//    printf("i=%d is %d\n",i,d_result[i]);
    x+=(double)numBlocks;
  }
}

//int main(void)??
int main(int argc, char *argv[]) {
  //change the params below later
  int numBlocks=512;
  clock_t start_time = clock();
  assert(argc==2);

  double stop = (double)atol(argv[1]);
  assert(stop >= 1.0);

  int *result = (int*)malloc(numBlocks*sizeof(int));
  for(int i=0;i<numBlocks;i++){
    result[i]=0;
  }


  int *d_result;
  hipMalloc((void**)&d_result, numBlocks*sizeof(int));
  hipMemcpy(d_result, result, numBlocks*sizeof(int), hipMemcpyHostToDevice);

  pin<<<numBlocks,1>>>(d_result,stop,numBlocks);

  //MPI_Reduce used to be here
  hipMemcpy(result, d_result, numBlocks*sizeof(int),hipMemcpyDeviceToHost);
  
  int pin=0;

  for(int i=0;i<numBlocks;i++){
    pin=(pin+result[i])%10000;
//    printf("result %d is %d.\n",i,result[i]);
  }
  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  printf("The PIN is %d (numBlocks = %d, time = %f sec.)\n", pin, numBlocks, time);
  hipFree(d_result);
  free(result);
  fflush(stdout);
  return 0;
}
