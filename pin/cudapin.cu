#include "hip/hip_runtime.h"
/* 
 * Some tests:
 * 1 -> 0
 * 2 -> 7080
 * 100 -> 0076
 *
 * 100000000 takes 4.2 seconds on my MacBook Air
 * 1000000000  takes 43 seconds on same machine
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
//int rank;
//int numprocs;
__global__ void pin(int *d_result, double stop){
  for(double x=(double)blockIdx.x; x<stop;x+=(double)gridDim.x){
    double tmp=sin(x);
    tmp=tmp*tmp;
    int z=(int) (tmp*10000.0);
    d_result[(int)x]=(d_result[(int)x]+z)%10000;
//    printf("x=%d is %f\n",x,d_result[(int)x]);
  }
}

//int main(void)??
int main(int argc, char *argv[]) {
  //change the params below later
  int numBlocks=100;
  clock_t start_time = clock();
  assert(argc==2);

  double stop = (double)atol(argv[1]);
  assert(stop >= 1.0);

  int *result = (int*)malloc(numBlocks*sizeof(int));
  for(int i=0;i<numBlocks;i++){
    result[i]=0;
  }


  int *d_result;
  hipMalloc((void**)&d_result, numBlocks*sizeof(int));
  hipMemcpy(d_result, &result, numBlocks*sizeof(int), hipMemcpyHostToDevice);

  pin<<<numBlocks,1>>>(d_result,stop);

  //MPI_Reduce used to be here
  hipMemcpy(result, d_result, sizeof(int),hipMemcpyDeviceToHost);
  
  int pin=0;

  for(int i=0;i<numBlocks;i++){
    pin=(pin+result[i])%10000;	
  }
  clock_t finish_time = clock();
  double time = (double)(finish_time-start_time)/CLOCKS_PER_SEC;
  printf("The PIN is %d (numBlocks = %d, time = %f sec.)\n", pin, numBlocks, time);
  hipFree(d_result);
  fflush(stdout);
  return 0;
}
